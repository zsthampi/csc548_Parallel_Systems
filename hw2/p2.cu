#include "hip/hip_runtime.h"
/* Program to compute Pi using the Leibniz formula */

#include <stdio.h>
#include <math.h>
#include "mytime.h"

#define THREADS 512
#define MAX_BLOCKS 4

__global__ void Leibniz(int *n, int *blocks, double *gsum) {
  __shared__ double sum[THREADS];
  int k;
  double power;
  
  sum[threadIdx.x] = 0.0;
  for (k = blockIdx.x * blockDim.x + threadIdx.x; k < *n; k += blockDim.x * *blocks) {
    if (k%2==0) {
      power = 1.0;
    } else {
      power = -1.0;
    }
    sum[threadIdx.x] += (power / (double)(2.0 * k + 1.0));
  }

  sum[threadIdx.x] *= 4.0;

  __syncthreads();

  // Block Reduction
  for (k = blockDim.x/2; k > 0; k >>= 1) {
    if (threadIdx.x < k) {
      sum[threadIdx.x] += sum[threadIdx.x + k];
    }
    __syncthreads();
  }

  // k = blockDim.x;
  // while (k>0) {
  //   if (k%2==0) {
  //     k = k/2;
  //   } else {
  //     k = (k+1)/2;
  //   }
  //   if (threadIdx.x < k && (threadIdx.x + k) < blockDim.x) {
  //     sum[threadIdx.x] += sum[threadIdx.x + k];
  //   }
  //   __syncthreads();
  // }

  if (threadIdx.x == 0 )
    gsum[blockIdx.x] = sum[threadIdx.x];
}

__global__ void global_reduce(int *n, int *blocks, double *gsum) {
  int k;
  __shared__ double sum[THREADS];

  sum[threadIdx.x] = gsum[threadIdx.x];
  __syncthreads();

  for (k = blockDim.x/2; k > 0 ; k >>= 1) {
    if (threadIdx.x < k) {
      sum[threadIdx.x] += sum[threadIdx.x + k];
    }
    __syncthreads();
  }

  // k = blockDim.x;
  // while (k>0) {
  //   if (k%2==0) {
  //     k = k/2;
  //   } else {
  //     k = (k+1)/2;
  //   }
  //   if (threadIdx.x < k && (threadIdx.x + k) < blockDim.x) {
  //     sum[threadIdx.x] += sum[threadIdx.x + k];
  //   }
  //   __syncthreads();
  // }

  if (threadIdx.x == 0)
    gsum[threadIdx.x] = sum[threadIdx.x];
}

int main(int argc, char *argv[]) {
  int n;
  int blocks = MAX_BLOCKS;
  // Device copy of number of intervals
  int *n_d, *blocks_d;
  double PI25DT = 3.141592653589793238462643;
  double pi;
  double mypi[THREADS * blocks];
  // Device copy of computed pi value
  double *mypi_d;
  struct timeval startwtime, endwtime, diffwtime;
  
  // Allocate memory
  hipMalloc((void **) &n_d, sizeof(int));
  hipMalloc((void **) &blocks_d, sizeof(int));
  hipMalloc((void **) &mypi_d, sizeof(double) * blocks);

  while (1) {
    printf("Enter the number of intervals: (0 quits) ");fflush(stdout);
    scanf("%d",&n);

    gettimeofday(&startwtime, NULL);
    if (n == 0)
      break;

    // Copy from Host to Device
    hipMemcpy(n_d,&n,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(blocks_d,&blocks,sizeof(int),hipMemcpyHostToDevice);

    Leibniz<<< blocks,THREADS >>>(n_d,blocks_d,mypi_d);

    // Copy from Device to Host
    // hipMemcpy(&mypi,mypi_d,sizeof(double) * THREADS * blocks,hipMemcpyDeviceToHost);

    global_reduce<<< 1,blocks >>>(n_d,blocks_d,mypi_d);
    hipMemcpy(&pi,mypi_d,sizeof(double),hipMemcpyDeviceToHost);

    hipMemcpy(&mypi,mypi_d,sizeof(double) * blocks,hipMemcpyDeviceToHost);
    for (int i=0; i<blocks; i++) {
      printf("%d : %lf \n",i,mypi[i]);
    }

    // pi = 0.0;
    // for (int i=0; i<THREADS; i++) {
    //   pi += mypi[i];
    // }

    gettimeofday(&endwtime, NULL);
    MINUS_UTIME(diffwtime, endwtime, startwtime);
    printf("pi is approximately %.16f, Error is %.16f\n",
	   pi, fabs(pi - PI25DT));
    printf("wall clock time = %d.%06d\n",
	   diffwtime.tv_sec, diffwtime.tv_usec);
  }

  // Free memory
  hipFree(n_d);
  hipFree(blocks_d);
  hipFree(mypi_d);
  return 0;
}
